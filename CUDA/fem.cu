
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <limits.h>

#define TEMP_BOT 0
#define TEMP_LEFT 0
#define TEMP_UP 200
#define TEMP_RIGHT 200
#define TEMP_BEGIN 20

#define ANIMATION_FRAME_DELAY 10
#define TIME 21
#define ALFA1 1
#define ALFA2 1

#define EPS 0.001

float delta;
float eps = INT_MAX;
long long unsigned int x_nodes_num;
long long unsigned int y_nodes_num;
long long unsigned int nodes_num;
FILE *fp;
FILE *gnuPlot;

long long unsigned int N_threads = 512;
long long unsigned int N_blocks;

float get_element(float *x, long long unsigned int  row_size, long long unsigned int  i, long long unsigned int  j)
{
    return x[i * row_size + j];
}

void set_element(float value, float *x, long long unsigned int row_size, long long unsigned int  i, long long unsigned int  j)
{
    x[i * row_size + j] = value;
}

void initAB(float *A, float *B, float *x)
{
    long long unsigned int i = 0, j = 0, offset = 0, step = 0, index = 0;
    // Заполнѝем нижний левый угол плаѝтины
    set_element(1, A, nodes_num, 0, 0);
    set_element(-1, A, nodes_num, 0, 1);
    B[0] = 0;
    // Заполнѝем верхний край плаѝтины
    offset = x_nodes_num * (y_nodes_num - 1);
    for (i = 0; i < x_nodes_num; i++)
    {
        set_element(1, A, nodes_num, i + offset, i + offset);
        B[i + offset] = TEMP_UP;
    }

    // Заполнѝем правый край плаѝтины
    offset = x_nodes_num - 1;
    index = offset;
    step = x_nodes_num;
    for (i = 0; i < y_nodes_num; i++)
    {
        set_element(1, A, nodes_num, index, index);
        B[index] = TEMP_RIGHT;
        index += step;
    }

    // Заполнѝем левый край плаѝтины
    offset = x_nodes_num;
    index = offset;
    step = x_nodes_num;
    for (i = 0; i < y_nodes_num - 2; i++)
    {
        if (i == 2)
        {
            set_element(1 + delta, A, nodes_num, index, index);
            set_element(-1, A, nodes_num, index, index + 1);
            B[index] = 0;
        }
        else
        {
            set_element(1, A, nodes_num, index, index);
            set_element(-1, A, nodes_num, index, index + 1);
            B[index] = 0;
        }
        index += step;
    }

    // Заполнѝем нижний край плаѝтины
    offset = 1;
    index = offset;
    step = 1;
    for (i = 0; i < x_nodes_num - 2; i++)
    {
        set_element(1, A, nodes_num, index, index);
        set_element(-1, A, nodes_num, index, index + x_nodes_num);
        B[index] = 0;
        index += step;
    }
    // Заполнѝем внутреннюю чаѝть плаѝтины
    offset = x_nodes_num + 1;
    step = x_nodes_num;
    for (i = 0; i < y_nodes_num - 2; i++)
    {
        for (j = 0; j < x_nodes_num - 2; j++)
        {
            index = offset + i * x_nodes_num + j;
            set_element(5, A, nodes_num, index, index);
            set_element(-1, A, nodes_num, index, index - 1);
            set_element(-1, A, nodes_num, index, index + 1);
            set_element(-1, A, nodes_num, index, index - x_nodes_num);
            set_element(-1, A, nodes_num, index, index + x_nodes_num);
            B[index] = x[index];
        }
    }
}

void initB(float *B, float *x)
{
    long long unsigned int i = 0, j = 0, offset = 0, index = 0, step = 0;
    // Заполнѝем внутреннюю чаѝть плаѝтины
    offset = x_nodes_num + 1;
    step = x_nodes_num;
    for (i = 0; i < y_nodes_num - 2; i++)
    {
        for (j = 0; j < x_nodes_num - 2; j++)
        {
            index = offset + i * x_nodes_num + j;
            B[index] = x[index];
        }
    }
}

void initX(float *x)
{
    for (long long unsigned int  i = 0; i < nodes_num; i++)
    {
        x[i] = (float)TEMP_BEGIN;
    }
}

void initX0(float *A, float *x, float *B)
{
    for (long long unsigned int  i = 0; i < nodes_num; i++)
    {
        float aii = get_element(A, nodes_num, i, i);
        x[i] = B[i] / aii;
    }
}

void printToFile(float *temp)
{
    for (int j = y_nodes_num - 1; j >= 0; j--)
    {
        for (int i = 0; i < x_nodes_num; i++)
        {
            fprintf(fp, "%.1f\t", temp[j * x_nodes_num + i]);
        }
        fprintf(fp, "\n");
    }
    fprintf(fp, "\n");
}

void zeroingA(float *A)
{
    long long unsigned int i = 0, j = 0;
    for (i = 0; i < nodes_num; i++)
    {
        for (j = 0; j < nodes_num; j++)
        {
            set_element(0, A, nodes_num, i, j);
        }
    }
}

void zeroingB(float *vectorB)
{
    long long unsigned int i = 0;
    for (i = 0; i < nodes_num; i++)
    {
        vectorB[i] = 0;
    }
}

__global__ void Solve(float *dA, float *dF, float *dX0, float *dX1, long long unsigned int N)
{
    float aa, sum = 0.;
    long long unsigned int t = blockIdx.x * blockDim.x + threadIdx.x;

    for (long long unsigned int j = 0; j < N; j++)
    {
        sum += dA[j + t * N] * dX0[j];
    }
    aa = dA[t + t * N];
    dX1[t] = dX0[t] + (dF[t] - sum) / aa;
}

__global__ void Eps(float *dX0, float *dX1, float *delta, long long unsigned int N)
{
    long long unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    delta[i] = fabs(dX0[i] - dX1[i]);
    dX0[i] = dX1[i];
}

int main(int argc, char **argv)
{
    if (argc != 3) {
        printf("Error data. Please, enter these aruments:\n1st argument - x_nodes;\n2nd argument - y_nodes.\n");
        return 1;
    }

    x_nodes_num = atoi(argv[1]);
    y_nodes_num = atoi(argv[2]);

    if (x_nodes_num == 0 || y_nodes_num == 0) {
        printf("Error data. Please, enter 2 integers.\n");
        return 1;
    }

    nodes_num = x_nodes_num * y_nodes_num;
    delta = 1;
    long long unsigned int  i = 0, j = 0;
    hipEvent_t GPUstart, GPUstop;
    float GPUtime = 0.0f;

    N_threads = 6;

    if ((nodes_num % N_threads) == 0)
    {
        N_blocks = (nodes_num / N_threads);
    }
    else
    {
        N_blocks = (nodes_num / N_threads) + 1;
    }

    printf("x_nodes_num = %llu\ny_nodes_num=%llu\nN_threads=%llu\nN_blocks=%llu\n", x_nodes_num, y_nodes_num, N_threads, N_blocks);

    dim3 Threads(N_threads);
    dim3 Blocks(N_blocks);

    hipEventCreate(&GPUstart);
	hipEventCreate(&GPUstop);

	hipEventRecord(GPUstart, 0);

    float *hA = (float *)calloc(nodes_num * nodes_num, sizeof(float));
    float *hF = (float *)calloc(nodes_num, sizeof(float));
    float *hX = (float *)calloc(nodes_num, sizeof(float));
    float *hX0 = (float *)calloc(nodes_num, sizeof(float));
    float *hX1 = (float *)calloc(nodes_num, sizeof(float));
    float *hDelta = (float *)calloc(nodes_num, sizeof(float));

    float *dA, *dF, *dX0, *dX1, *dDelta;

    hipMalloc((void **)&dA, nodes_num * nodes_num * sizeof(float)); // матрица A
    hipMalloc((void **)&dF, nodes_num * sizeof(float));             // правая часть F
    hipMalloc((void **)&dX0, nodes_num * sizeof(float));            // решение X(n)
    hipMalloc((void **)&dX1, nodes_num * sizeof(float));            // решение X(n+1)
    hipMalloc((void **)&dDelta, nodes_num * sizeof(float));         // разница |X(n+1)- X(n)|

    fp = fopen("result_CUDA.txt", "w");
    if (fp == NULL)
    {
        printf("Open failed\n");
        return -1;
    }

    long long unsigned int  times = (TIME - 1) / delta;
    printf("Times: %llu\n", times);

    initX(hX);
    initAB(hA, hF, hX);
    initX0(hA, hX0, hF);
    hipMemcpy(dA, hA, nodes_num * nodes_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dX0, hX0, nodes_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dF, hF, nodes_num * sizeof(float), hipMemcpyHostToDevice);

    for (i = 1; i <= times; i++)
    {
        fprintf(fp, "%llu sec\n", i);
        printf("%llu\n", i);
        long long unsigned int k = 0;
        eps = INT_MAX;
        while (eps > EPS)
        {
            k++;
            Solve<<<Blocks, Threads>>>(dA, dF, dX0, dX1, nodes_num);
            Eps<<<Blocks, Threads>>>(dX0, dX1, dDelta, nodes_num);

            hipMemcpy(hX1, dX1, nodes_num * sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(hDelta, dDelta, nodes_num * sizeof(float), hipMemcpyDeviceToHost);
            eps = 0.;
            for (j = 0; j < nodes_num; j++)
            {
                eps += hDelta[j];
            }
            eps = eps / nodes_num;
            // printf("\n Eps[%i]=%e ", k, eps);
            // cudaMemcpy(dX0, dX1, nodes_num * sizeof(float), cudaMemcpyDeviceToDevice);
        }

        printToFile(hX1);
        hipMemcpy(hX, dX1, nodes_num * sizeof(float), hipMemcpyDeviceToHost);
        initB(hF, hX);
        hipMemcpy(dF, hF, nodes_num * sizeof(float), hipMemcpyHostToDevice);
        initX0(hA, hX0, hF);
        hipMemcpy(dX0, hX0, nodes_num * sizeof(float), hipMemcpyHostToDevice);
    }

    hipEventRecord(GPUstop, 0);
	hipEventSynchronize(GPUstop);

	hipEventElapsedTime(&GPUtime, GPUstart, GPUstop);
	printf("GPU time : %.3f ms\n", GPUtime);


    fclose(fp);

    // Оѝвобождение памѝи
    free(hA);
    free(hF);
    free(hX);
    free(hX0);
    free(hX1);
    free(hDelta);

    hipFree(dA);
    hipFree(dF);
    hipFree(dX0);
    hipFree(dX1);
    hipFree(dDelta);
}
